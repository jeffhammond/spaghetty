/*
 * My heavily modified version of the NVIDIA transpose code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <transpose_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
extern "C" void computeGold( float* reference, float* idata, 
                         const unsigned int size_a, const unsigned int size_b,
                         const unsigned int size_c, const unsigned int size_d );

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // size of the matrix
#ifdef __DEVICE_EMULATION__
    const unsigned int size_a = 4;
    const unsigned int size_b = 4;
    const unsigned int size_c = 4;
    const unsigned int size_d = 4;
#else
    const unsigned int size_a = 4;
    const unsigned int size_b = 4;
    const unsigned int size_c = 4;
    const unsigned int size_d = 4;
//    const unsigned int size_a = 64;
//    const unsigned int size_b = 64;
//    const unsigned int size_c = 64;
//    const unsigned int size_d = 64;
#endif
    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * size_a * size_b * size_c * size_d;
    
    unsigned int timer;
    cutCreateTimer(&timer);

    CUT_DEVICE_INIT(argc, argv);

    // allocate host memory
    float* h_idata = (float*) malloc(mem_size);
    // initalize the memory
    unsigned int num=0;
    for( unsigned int i = 0; i < (size_a); ++i){
      for( unsigned int j = 0; j < (size_b); ++j){
        for( unsigned int k = 0; k < (size_c); ++k){
          for( unsigned int l = 0; l < (size_d); ++l){
            h_idata[num++] = (float) (i + j*10 + k*100 + l*1000);
          }
        }
      }
    }

    // allocate device memory
    float* d_idata;
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));

    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // setup execution parameters
    dim3 dimGrid(1,1,1);
    dim3 dimBlock(1,1,1);

    // warmup so we don't time CUDA startup
    transpose<<< dimGrid, dimBlock >>>(d_odata, d_idata, size_a, size_b, size_c, size_d);
    
    int numIterations = 1;

    printf("Transposing a %d by %d by %d by %d matrix of floats...\n", size_a, size_b, size_c, size_d);

    // execute the kernel
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
        transpose<<< dimGrid, dimBlock >>>(d_odata, d_idata, size_a, size_b, size_c, size_d);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float gpuTime = cutGetTimerValue(timer);

    printf("GPU transpose average time: %0.3f ms\n", gpuTime / numIterations);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    float* h_odata = (float*) malloc(mem_size);
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );

    // compute reference solution
    float* reference = (float*) malloc( mem_size);

    // execute the kernel
    cutResetTimer(timer);
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i)
    {
         computeGold( reference, h_idata, size_a, size_b, size_c, size_d);
    }
    cutStopTimer(timer);
    float cpuTime = cutGetTimerValue(timer);

    printf("CPU transpose average time: %0.3f ms\n", cpuTime / numIterations);

    // check result

    printf("==================================================================\n");
    printf("                    Initial      Reference     OutputData\n");
    printf("==================================================================\n");
    for( unsigned int i = 0; i < (size_a); ++i){
      for( unsigned int j = 0; j < (size_b); ++j){
        for( unsigned int k = 0; k < (size_c); ++k){
          for( unsigned int l = 0; l < (size_d); ++l){
            printf("%3d %3d %3d %3d", i, j, k, l);
            printf("%14.7f",h_idata[i+size_b*(j+size_c*(k+size_d*l))]);
            printf("%14.7f",reference[i+size_b*(j+size_c*(k+size_d*l))]);
            printf("%14.7f\n",h_odata[i+size_b*(j+size_c*(k+size_d*l))]);
          }
        }
      }
    }
    CUTBoolean res = cutComparef( reference, h_odata, size_a * size_b * size_c * size_d);
    printf("==================================================================\n");
    printf(    "Test %s\n", (1    == res)    ? "PASSED" : "FAILED");
    printf("==================================================================\n");

    printf("GPU transpose average time: %0.3f ms\n", gpuTime / numIterations);
    printf("CPU transpose average time: %0.3f ms\n", cpuTime / numIterations);

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free( reference);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
}
