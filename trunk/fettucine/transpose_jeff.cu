/*
 * My heavily modified version of the NVIDIA transpose code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil.h>
#include "/opt/nvidia/cuda/common/inc/cutil.h"

// includes, kernels
#include <transpose_kernel_jeff.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
extern "C" void computeGold( float* reference, float* idata, 
                         const unsigned int size_a, const unsigned int size_b,
                         const unsigned int size_c, const unsigned int size_d );

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // number of runs to average timing over
    int numIterations = 8;
    // size of the matrix
#ifdef __DEVICE_EMULATION__
    const unsigned int size_a = 4;
    const unsigned int size_b = 4;
    const unsigned int size_c = 4;
    const unsigned int size_d = 4;
#else
    const unsigned int size = 8;
    const unsigned int size_a = size;
    const unsigned int size_b = size;
    const unsigned int size_c = size;
    const unsigned int size_d = size;
#endif
    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * size_a * size_b * size_c * size_d;
    
    unsigned int timer;
    cutCreateTimer(&timer);

    CUT_DEVICE_INIT(argc, argv);

    // allocate host memory
    float* h_idata = (float*) malloc(mem_size);
    // initalize the memory
    unsigned int num=0;
    for( unsigned int i = 0; i < (size_a); ++i){
      for( unsigned int j = 0; j < (size_b); ++j){
        for( unsigned int k = 0; k < (size_c); ++k){
          for( unsigned int l = 0; l < (size_d); ++l){
            h_idata[num++] = (float) (i + j*10 + k*100 + l*1000);
          }
        }
      }
    }

    // allocate device memory
    float* d_idata;
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));

    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    printf("Transposing a %d by %d by %d by %d matrix of floats...\n", size_a, size_b, size_c, size_d);

    // setup execution parameters
    dim3 dimBlock(8,8,8);
    dim3 dimGrid(size_a/dimBlock.x, size_b/dimBlock.y, size_c/dimBlock.z);

    // warmup so we don't time CUDA startup
    transpose_jeff<<< dimGrid, dimBlock >>>(d_odata, d_idata, size_a, size_b, size_c, size_d);

    // execute the kernel
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i){
        transpose_jeff<<< dimGrid, dimBlock >>>(d_odata, d_idata, size_a, size_b, size_c, size_d);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float gpuTime = cutGetTimerValue(timer);

    printf("GPU transpose average time: %0.3f ms\n", gpuTime / numIterations);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    float* h_odata = (float*) malloc(mem_size);
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );

    // compute reference solution
    float* reference = (float*) malloc( mem_size);

    // execute the kernel
    cutResetTimer(timer);
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i){
         computeGold( reference, h_idata, size_a, size_b, size_c, size_d);
    }
    cutStopTimer(timer);
    float cpuTime = cutGetTimerValue(timer);

    printf("CPU transpose average time: %0.3f ms\n", cpuTime / numIterations);

    // check result

    if((size_a * size_b * size_c * size_d)<10001){
    printf("==================================================================\n");
    printf("                    Initial      Reference     OutputData\n");
    printf("==================================================================\n");
    for( unsigned int i = 0; i < (size_a); ++i){
      for( unsigned int j = 0; j < (size_b); ++j){
        for( unsigned int k = 0; k < (size_c); ++k){
          for( unsigned int l = 0; l < (size_d); ++l){
            printf("%3d %3d %3d %3d", i, j, k, l);
            printf("%14.7f",h_idata[i+size_b*(j+size_c*(k+size_d*l))]);
            printf("%14.7f",reference[i+size_b*(j+size_c*(k+size_d*l))]);
            printf("%14.7f\n",h_odata[i+size_b*(j+size_c*(k+size_d*l))]);
          }
        }
      }
    }
    }
    CUTBoolean res = cutComparef( reference, h_odata, size_a * size_b * size_c * size_d);
    printf("==================================================================\n");
    printf(    "Test %s\n", (1    == res)    ? "PASSED" : "FAILED");
    printf("==================================================================\n");

    printf("GPU transpose average time: %0.3f ms\n", gpuTime / numIterations);
    printf("CPU transpose average time: %0.3f ms\n", cpuTime / numIterations);
    printf("Averaged over %d runs\n", numIterations);

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free( reference);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
}
