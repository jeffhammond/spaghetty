#include "hip/hip_runtime.h"
/*
 * My heavily modified version of the NVIDIA transpose code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil.h>
#include "/opt/nvidia/cuda/common/inc/cutil.h"

// includes, kernels
#include <transpose_kernel.cu>

// Thread block size
//#define BLOCK_SIZE 4

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
extern "C" void computeGold( float* refdata, float* idata, 
                         const unsigned int size_a, const unsigned int size_b,
                         const unsigned int size_c, const unsigned int size_d );

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int temp;
    //printf("argc = %d\n",argc);
    //printf("argv[1] = %s\n",argv[1]);
    if (argc > 1){
      temp = atoi(argv[1]);
      printf("size = %d\n",temp);
    } else {
      temp = 4;
    }

    // number of runs to average timing over
    int numIterations = 10;
    // size of the matrix
#ifdef __DEVICE_EMULATION__
    const unsigned int size_a = 4;
    const unsigned int size_b = 4;
    const unsigned int size_c = 4;
    const unsigned int size_d = 4;
#else
    unsigned int size = temp;
    const unsigned int size_a = size;
    const unsigned int size_b = size;
    const unsigned int size_c = size;
    const unsigned int size_d = size;
#endif

    // size of memory required to store the matrix
    const unsigned int mem_size = sizeof(float) * size_a * size_b * size_c * size_d;
    
    unsigned int timer;
    cutCreateTimer(&timer);

    CUT_DEVICE_INIT(argc, argv);

    // allocate host memory
    float* h_idata = (float*) malloc(mem_size);
    // initalize the memory
    unsigned int num=0;
    for( unsigned int i = 0; i < (size_a); ++i){
      for( unsigned int j = 0; j < (size_b); ++j){
        for( unsigned int k = 0; k < (size_c); ++k){
          for( unsigned int l = 0; l < (size_d); ++l){
            h_idata[num++] = (float) (i + j*10 + k*100 + l*1000);
          }
        }
      }
    }

    // to zero the device memory
    float* h_zero = (float*) calloc(mem_size,sizeof(float));

    // allocate device memory
    float* d_idata;
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));

    // copy host memory to device (zero output array)
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy( d_odata, h_zero, mem_size,
                                hipMemcpyHostToDevice) );

    printf("Transposing a %d by %d by %d by %d matrix of floats...\n", size_a, size_b, size_c, size_d);

    // setup execution parameters
    dim3 dimGrid((int)ceil((double)size_c/(double)BLOCK_SIZE),(int)ceil((double)size_d/(double)BLOCK_SIZE),1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    // warmup so we don't time CUDA startup
    transpose<<< dimGrid, dimBlock >>>(d_odata, d_idata, size_a, size_b, size_c, size_d);

    // execute the kernel
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i){
        transpose<<< dimGrid, dimBlock >>>(d_odata, d_idata, size_a, size_b, size_c, size_d);
    }
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float gpuTime = cutGetTimerValue(timer);

    printf("GPU transpose average time: %0.3f ms\n", gpuTime / numIterations);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    float* h_odata = (float*) malloc(mem_size);
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, mem_size,
                                hipMemcpyDeviceToHost) );

    // compute refdata solution
    float* refdata = (float*) malloc( mem_size);

    // execute the kernel
    cutResetTimer(timer);
    cutStartTimer(timer);
    for (int i = 0; i < numIterations; ++i){
         computeGold( refdata, h_idata, size_a, size_b, size_c, size_d);
    }
    cutStopTimer(timer);
    float cpuTime = cutGetTimerValue(timer);

    printf("CPU transpose average time: %0.3f ms\n", cpuTime / numIterations);

    // check result

    if((size_a * size_b * size_c * size_d)<100000){
    unsigned int offset;
    printf("==================================================================\n");
    printf("                    Initial      Reference     OutputData\n");
    printf("==================================================================\n");
    for( unsigned int i = 0; i < (size_a); ++i){
      for( unsigned int j = 0; j < (size_b); ++j){
        for( unsigned int k = 0; k < (size_c); ++k){
          for( unsigned int l = 0; l < (size_d); ++l){
            offset = i+size_b*(j+size_c*(k+size_d*l));
            if (h_odata[offset] != refdata[offset]){
              printf("%3d %3d %3d %3d", i, j, k, l);
              printf("%14.7f",h_idata[offset]);
              printf("%14.7f",refdata[offset]);
              printf("%14.7f",h_odata[offset]);
              //printf("%7d",(int)(h_odata[offset] == refdata[offset]));
              printf("\n");
            }
          }
        }
      }
    }
    }
    CUTBoolean res = cutComparef( refdata, h_odata, size_a * size_b * size_c * size_d);
    printf("==================================================================\n");
    printf(    "Test %s\n", (1    == res)    ? "PASSED" : "FAILED");
    printf("==================================================================\n");

    printf("GPU transpose average time: %0.3f ms\n", gpuTime / numIterations);
    printf("CPU transpose average time: %0.3f ms\n", cpuTime / numIterations);
    printf("Averaged over %d runs\n", numIterations);

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free( refdata);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
}
