#include "hip/hip_runtime.h"
#ifndef _TRANSPOSE_KERNEL_H_
#define _TRANSPOSE_KERNEL_H_

#define BLOCK_SIZE 16

__global__ void transpose(float *odata, float *idata,
                          const unsigned int dim_1, const unsigned int dim_2,
                          const unsigned int dim_3, const unsigned int dim_4)
{
	__shared__ float block[BLOCK_SIZE][BLOCK_SIZE+1];
	
	// read the matrix tile into shared memory
	unsigned int xIndex = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_SIZE + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_SIZE + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}


__global__ void transpose_naive(float *odata, float* idata,
                                const unsigned int dim_1, const unsigned int dim_2,
                                const unsigned int dim_3, const unsigned int dim_4)
{
   unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
   
   if (xIndex < width && yIndex < height)
   {
       unsigned int index_in  = xIndex + width * yIndex;
       unsigned int index_out = yIndex + height * xIndex;
       odata[index_out] = idata[index_in]; 
   }
}

#endif // _TRANSPOSE_KERNEL_H_
